#include "hip/hip_runtime.h"
#include <iostream>

#include "Hub.h"
#include "Random.h"
#include "CustomException.h"
#include "ConvolutionalLayer.h"
#include "hip/hip_runtime.h"
#include ""

#define BLOCK_SIZE 256
#define DOUBLE_BLOCK_SIZE 32
#define LearningRate 0.0005f

texture<float, 2> MatrixesRef;
texture<float, 2> FiltersRef;
texture<float, 2> OutputsRef;

__global__ void cuda_convolve(float* feature_map, float* biases, const int inp_cols, const int fm_cols, const int fm_rows, size_t fm_pitch, const int fm_depth, const int filter_size)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int z = blockDim.z * blockIdx.z + threadIdx.z;

	if (x < fm_cols && y < fm_rows && z < fm_depth)
	{
		float* feature_map_matrix_start = (float*)((char*)feature_map + z * fm_pitch);
		int feature_map_position = y * fm_cols + x;
		float sum = biases[z];
		for (int i = 0; i < filter_size; i++)
		{
			for (int j = 0; j < filter_size; j++)
			{
				int matrix_position = (y + i) * inp_cols + (x + j);
				int filter_position = i * filter_size + j;
				sum += tex2D(MatrixesRef, matrix_position, blockIdx.z) * tex2D(FiltersRef, filter_position, threadIdx.z);
			}
		}
		feature_map_matrix_start[feature_map_position] = __max(sum, 0);
	}
}

__global__ void cuda_cross_correlation(float* prev_gradients, const int prev_gr_cols, const int prev_gr_rows, size_t prev_gr_pitch, const int gr_cols, const int gr_rows, const int filter_size, const int filters_count, const int padding)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int z = blockIdx.z;

	if (x < prev_gr_cols && y < prev_gr_rows)
	{
		float* prev_gradient_matrix_start = (float*)((char*)prev_gradients + z * prev_gr_pitch);
		int prev_gradient_position = y * prev_gr_cols + x;
		float delta_sum = 0.0f;
		prev_gradient_matrix_start[prev_gradient_position] = 0.0f;

		for (int i = 0; i < filter_size; i++)
		{
			for (int j = 0; j < filter_size; j++)
			{
				int gr_y_pos = y + i - padding;
				int gr_x_pos = x + j - padding;
				float is_inside = (gr_y_pos >= 0 && gr_y_pos < gr_rows && gr_x_pos >= 0 && gr_x_pos < gr_cols);

				int matrix_position = gr_y_pos * gr_cols + gr_x_pos;
				int filter_position = (padding - i) * filter_size + (padding - j);

				for (int l = 0; l < filters_count; l++)
				{
					delta_sum += is_inside * tex2D(MatrixesRef, matrix_position, z * filters_count + l) * tex2D(FiltersRef, filter_position, l);
				}
			}
		}
		prev_gradient_matrix_start[prev_gradient_position] += delta_sum;
	}
}

__global__ void cuda_correct_filters(float* filters, const int fl_cols, const int fl_rows, size_t fl_pitch, const int gr_cols, const int gr_rows, const int in_cols, const int in_count, const int fl_count)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int z = blockIdx.z;

	if (x < gr_cols && y < gr_rows)
	{
		float* filter_matrix_start = (float*)((char*)filters + z * fl_pitch);
		int filter_position = y * fl_cols + x;
		float delta_sum = 0.0f;
		for (int l = 0; l < in_count; l++)
		{
			int gr_num = l * fl_count + z;
			int gr_position = y * gr_cols + x;
			for (int i = 0; i < fl_rows; i++)
			{
				for (int j = 0; j < fl_cols; j++)
				{
					int matrix_position = (y + i) * in_cols + (x + j);
					delta_sum += (tex2D(OutputsRef, gr_position, gr_num) > 0.0f)* tex2D(MatrixesRef, matrix_position, l)* tex2D(FiltersRef, gr_position, gr_num);
				}
			}
		}
		filter_matrix_start[filter_position] -= LearningRate * delta_sum;
	}
}

__global__ void cuda_correct_biases(float* biases, const int b_size, const int gr_cols, const int gr_rows)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x < b_size)
	{
		float sum = 0.0f;
		for (int i = 0; i < gr_rows; i++)
		{
			for (int j = 0; j < gr_cols; j++)
			{
				sum = tex2D(FiltersRef, i * gr_cols + j, x);
			}
		}
		biases[x] -= LearningRate * sum;
	}
}

ConvolutionalLayer::ConvolutionalLayer(const int filters_size, const int filters_count, const int outputs_size, const int outputs_depth, Hub& params_storage) {
	
	filters_device = Tensor(filters_size, filters_size, filters_count);
	gradients_device = Tensor(outputs_size, outputs_size, outputs_depth);
	hipMallocPitch((void**)&gradients_device.data, &gradients_device.pitch, gradients_device.matrixes_size * sizeof(float), gradients_device.depth);

	outputs_devices = Tensor(outputs_size, outputs_size, filters_count);
	hipMallocPitch((void**)&outputs_devices.data, &outputs_devices.pitch, outputs_devices.matrixes_size * sizeof(float), outputs_devices.depth);

	if (params_storage.get_status() == Status::Training)
	{
		filters_device.data = set_normal_random(filters_size * filters_size, filters_count, filters_device.pitch);
		biases_device = set_repeatable_values(outputs_depth, 0.01f);
	}
	else
	{
		params_storage.get_params(filters_device);
		biases_device = params_storage.get_params(outputs_depth);
	}
}

Tensor& ConvolutionalLayer::forward(Tensor& input_matrixes) {
	
	inputs_device = input_matrixes;

	hipBindTexture2D(0, MatrixesRef, inputs_device.data, MatrixesRef.channelDesc, inputs_device.matrixes_size, inputs_device.depth, inputs_device.pitch);
	hipBindTexture2D(0, FiltersRef, filters_device.data, FiltersRef.channelDesc, filters_device.matrixes_size, filters_device.depth, filters_device.pitch);

	dim3 threadsPerBlock = dim3(10, 10, filters_device.depth);
	dim3 blocksPerGrid = dim3(outputs_devices.cols_count / 10 + (outputs_devices.cols_count % 10 == 0 ? 0 : 1),
		outputs_devices.rows_count / 10 + (outputs_devices.rows_count % 10 == 0 ? 0 : 1), inputs_device.depth);

	cuda_convolve << <blocksPerGrid, threadsPerBlock >> > (outputs_devices.data, biases_device, inputs_device.cols_count, outputs_devices.cols_count, outputs_devices.rows_count, outputs_devices.pitch, outputs_devices.depth, filters_device.cols_count);
	cudacall(hipGetLastError());

	hipUnbindTexture(MatrixesRef);
	hipUnbindTexture(FiltersRef);

	return outputs_devices;
}

void ConvolutionalLayer::backward(Tensor& prev_gradient_matrixes) {

	hipBindTexture2D(0, FiltersRef, filters_device.data, FiltersRef.channelDesc, filters_device.matrixes_size, filters_device.depth, filters_device.pitch);
	hipBindTexture2D(0, MatrixesRef, gradients_device.data, MatrixesRef.channelDesc, gradients_device.matrixes_size, gradients_device.depth, gradients_device.pitch);

	dim3 threadsPerBlock = dim3(DOUBLE_BLOCK_SIZE, DOUBLE_BLOCK_SIZE, 1);
	dim3 blocksPerGrid = dim3(prev_gradient_matrixes.cols_count / DOUBLE_BLOCK_SIZE + (prev_gradient_matrixes.cols_count % DOUBLE_BLOCK_SIZE == 0 ? 0 : 1),
		prev_gradient_matrixes.rows_count / DOUBLE_BLOCK_SIZE + (prev_gradient_matrixes.rows_count % DOUBLE_BLOCK_SIZE == 0 ? 0 : 1), prev_gradient_matrixes.depth);
	
	cuda_cross_correlation << <blocksPerGrid, threadsPerBlock >> > (prev_gradient_matrixes.data, prev_gradient_matrixes.cols_count, prev_gradient_matrixes.rows_count, prev_gradient_matrixes.pitch, gradients_device.cols_count, gradients_device.rows_count, filters_device.cols_count, filters_device.depth, filters_device.cols_count - 1);
	cudacall(hipGetLastError());

	hipUnbindTexture(FiltersRef);
	hipUnbindTexture(MatrixesRef);
}

void ConvolutionalLayer::correct() {

	hipBindTexture2D(0, FiltersRef, gradients_device.data, FiltersRef.channelDesc, gradients_device.matrixes_size, gradients_device.depth, gradients_device.pitch);
	hipBindTexture2D(0, MatrixesRef, inputs_device.data, MatrixesRef.channelDesc, inputs_device.matrixes_size, inputs_device.depth, inputs_device.pitch);
	hipBindTexture2D(0, OutputsRef, outputs_devices.data, OutputsRef.channelDesc, outputs_devices.matrixes_size, outputs_devices.depth, outputs_devices.pitch);
	
	dim3 threadsPerBlock = dim3(DOUBLE_BLOCK_SIZE, DOUBLE_BLOCK_SIZE, 1);
	dim3 blocksPerGrid = dim3(gradients_device.cols_count / DOUBLE_BLOCK_SIZE + (gradients_device.cols_count % DOUBLE_BLOCK_SIZE == 0 ? 0 : 1),
		gradients_device.rows_count / DOUBLE_BLOCK_SIZE + (gradients_device.rows_count % DOUBLE_BLOCK_SIZE == 0 ? 0 : 1), filters_device.depth);
	
	cuda_correct_filters << <blocksPerGrid, threadsPerBlock >> > (filters_device.data, filters_device.cols_count, filters_device.rows_count, filters_device.pitch, gradients_device.cols_count, gradients_device.rows_count, inputs_device.rows_count, inputs_device.depth, filters_device.depth);
	cudacall(hipGetLastError());

	threadsPerBlock = BLOCK_SIZE;
	blocksPerGrid = dim3(gradients_device.depth / BLOCK_SIZE + (gradients_device.depth % BLOCK_SIZE == 0 ? 0 : 1));
	
	cuda_correct_biases << <blocksPerGrid, threadsPerBlock >> > (biases_device, gradients_device.depth, gradients_device.cols_count, gradients_device.rows_count);
	cudacall(hipGetLastError());

	hipUnbindTexture(FiltersRef);
	hipUnbindTexture(MatrixesRef);
	hipUnbindTexture(OutputsRef);
}

void ConvolutionalLayer::save_params(Hub& params_storage) {

	params_storage.set_params(filters_device);
	params_storage.set_params(biases_device, outputs_devices.depth);
}

void ConvolutionalLayer::freeMemory() {

	hipFree(filters_device.data);
	hipFree(gradients_device.data);
	hipFree(inputs_device.data);
	hipFree(biases_device);
}