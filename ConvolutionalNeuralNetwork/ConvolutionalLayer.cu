#include "hip/hip_runtime.h"
#include <iostream>

#include "Hub.h"
#include "Random.h"
#include "CustomException.h"
#include "ConvolutionalLayer.h"
#include "hip/hip_runtime.h"
#include ""

#define BLOCK_SIZE 256
#define DOUBLE_BLOCK_SIZE 32
#define LearningRate 0.0005f
#define MAX_ELEMENTS_COUNT 1024
#define SUM_STRIDE 512

texture<float, 2> MatrixesRef;
texture<float, 2> FiltersRef;
texture<float, 2> OutputsRef;

__global__ void cuda_convolve(float* feature_map, float* biases, const int inp_cols, const int inp_depth, const int fm_cols, const int fm_rows, size_t fm_pitch, const int filter_size)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int z = blockIdx.z;

	if (x < fm_cols && y < fm_rows)
	{
		float* feature_map_matrix_start = (float*)((char*)feature_map + z * fm_pitch);
		int feature_map_position = y * fm_cols + x;
		float sum = biases[z];
		for (int i = 0; i < filter_size; i++)
		{
			for (int j = 0; j < filter_size; j++)
			{
				int matrix_position = (y + i) * inp_cols + (x + j);
				int filter_position = i * filter_size + j;
				for (int l = 0; l < inp_depth; l++)
					sum += tex2D(MatrixesRef, matrix_position, l) * tex2D(FiltersRef, filter_position, z * inp_depth + l);
			}
		}
		feature_map_matrix_start[feature_map_position] = __max(sum, 0);
	}
}

__global__ void cuda_cross_correlation(float* prev_gradients, const int prev_gr_cols, const int prev_gr_rows, size_t prev_gr_pitch, const int gr_cols, const int gr_rows, const int filter_size, const int filters_count, const int padding)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int z = blockIdx.z;

	if (x < prev_gr_cols && y < prev_gr_rows)
	{
		float* prev_gradient_matrix_start = (float*)((char*)prev_gradients + z * prev_gr_pitch);
		int prev_gradient_position = y * prev_gr_cols + x;
		float delta_sum = 0.0f;
		prev_gradient_matrix_start[prev_gradient_position] = 0.0f;

		for (int i = 0; i < filter_size; i++)
		{
			for (int j = 0; j < filter_size; j++)
			{
				int gr_y_pos = y + i - padding;
				int gr_x_pos = x + j - padding;
				float is_inside = (gr_y_pos >= 0 && gr_y_pos < gr_rows && gr_x_pos >= 0 && gr_x_pos < gr_cols);

				int matrix_position = gr_y_pos * gr_cols + gr_x_pos;
				int filter_position = (padding - i) * filter_size + (padding - j);

				for (int l = 0; l < filters_count; l++)
				{
					delta_sum += is_inside * tex2D(MatrixesRef, matrix_position, l) * tex2D(FiltersRef, filter_position, l * filters_count + z);
				}
			}
		}
		prev_gradient_matrix_start[prev_gradient_position] += delta_sum;
	}
}

__global__ void cuda_correct_filters(float* filters, const int fl_size, size_t fl_pitch, const int gr_cols, const int gr_rows, const int gr_count, const int in_cols, const int in_count)
{
	int x = threadIdx.x;
	int y = threadIdx.y;
	int x_block = blockIdx.x;
	int y_block = blockIdx.y;
	int z_block = blockIdx.z;

	int gr_num = z_block / in_count;
	int in_num = z_block / gr_count;

	extern __shared__ float s_deltas[];
	s_deltas[y * blockDim.x + x] = 0.0f;

	int gr_position = y * gr_rows + x;
	int filter_position = y_block * fl_size + x_block;

	if (x < gr_cols && y < gr_rows)
	{
		int matrix_position = (y + y_block) * in_cols + (x + x_block);
		float* filter_matrix_start = (float*)((char*)filters + z_block * fl_pitch);

		s_deltas[gr_position] = (tex2D(OutputsRef, gr_position, gr_num) > 0.0f)* tex2D(MatrixesRef, matrix_position, in_num)* tex2D(FiltersRef, gr_position, gr_num);
	}

	__syncthreads();
	
	for (unsigned int s = SUM_STRIDE; s > 0; s >>= 1) {
		if (gr_position < s)
			s_deltas[gr_position] += s_deltas[gr_position + s];
		__syncthreads();
	}
	if (gr_position == 0)
	{
		float* filter_matrix_start = (float*)((char*)filters + z_block * fl_pitch);
		filter_matrix_start[filter_position] -= LearningRate * s_deltas[0];
	}
}

__global__ void cuda_correct_biases(float* biases, const int b_size, const int gr_cols, const int gr_rows)
{
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x < b_size)
	{
		float sum = 0.0f;
		for (int i = 0; i < gr_rows; i++)
		{
			for (int j = 0; j < gr_cols; j++)
			{
				sum += tex2D(FiltersRef, i * gr_cols + j, x);
			}
		}
		biases[x] -= LearningRate * sum;
	}
}

ConvolutionalLayer::ConvolutionalLayer(const int filters_size, const int filters_count, const int inputs_depth, const int outputs_size, Hub& params_storage) {
	
	int filter_depth = inputs_depth * filters_count;
	filters_device = Tensor(filters_size, filters_size, filter_depth);
	gradients_device = Tensor(outputs_size, outputs_size, filters_count);
	hipMallocPitch((void**)&gradients_device.data, &gradients_device.pitch, gradients_device.matrixes_size * sizeof(float), gradients_device.depth);

	outputs_devices = Tensor(outputs_size, outputs_size, filters_count);
	hipMallocPitch((void**)&outputs_devices.data, &outputs_devices.pitch, outputs_devices.matrixes_size * sizeof(float), outputs_devices.depth);

	if (params_storage.get_status() == Status::Training)
	{
		filters_device.data = set_normal_random(filters_size * filters_size, filter_depth, filters_device.pitch);
		biases_device = set_repeatable_values(filters_count, 0.01f);
	}
	else
	{
		params_storage.get_params(filters_device);
		biases_device = params_storage.get_params(filters_count);
	}
}

Tensor& ConvolutionalLayer::forward(Tensor& input_matrixes) {
	
	inputs_device = input_matrixes;

	hipBindTexture2D(0, MatrixesRef, inputs_device.data, MatrixesRef.channelDesc, inputs_device.matrixes_size, inputs_device.depth, inputs_device.pitch);
	hipBindTexture2D(0, FiltersRef, filters_device.data, FiltersRef.channelDesc, filters_device.matrixes_size, filters_device.depth, filters_device.pitch);

	dim3 threadsPerBlock = dim3(DOUBLE_BLOCK_SIZE, DOUBLE_BLOCK_SIZE, 1);
	dim3 blocksPerGrid = dim3(outputs_devices.cols_count / DOUBLE_BLOCK_SIZE + (outputs_devices.cols_count % DOUBLE_BLOCK_SIZE == 0 ? 0 : 1),
		outputs_devices.rows_count / DOUBLE_BLOCK_SIZE + (outputs_devices.rows_count % DOUBLE_BLOCK_SIZE == 0 ? 0 : 1), outputs_devices.depth);

	cuda_convolve << <blocksPerGrid, threadsPerBlock >> > (outputs_devices.data, biases_device, inputs_device.cols_count, inputs_device.depth, outputs_devices.cols_count, outputs_devices.rows_count, outputs_devices.pitch, filters_device.cols_count);
	cudacall(hipGetLastError());

	hipUnbindTexture(MatrixesRef);
	hipUnbindTexture(FiltersRef);

	return outputs_devices;
}

void ConvolutionalLayer::backward(Tensor& prev_gradient_matrixes) {

	hipBindTexture2D(0, FiltersRef, filters_device.data, FiltersRef.channelDesc, filters_device.matrixes_size, filters_device.depth, filters_device.pitch);
	hipBindTexture2D(0, MatrixesRef, gradients_device.data, MatrixesRef.channelDesc, gradients_device.matrixes_size, gradients_device.depth, gradients_device.pitch);

	dim3 threadsPerBlock = dim3(DOUBLE_BLOCK_SIZE, DOUBLE_BLOCK_SIZE, 1);
	dim3 blocksPerGrid = dim3(prev_gradient_matrixes.cols_count / DOUBLE_BLOCK_SIZE + (prev_gradient_matrixes.cols_count % DOUBLE_BLOCK_SIZE == 0 ? 0 : 1),
		prev_gradient_matrixes.rows_count / DOUBLE_BLOCK_SIZE + (prev_gradient_matrixes.rows_count % DOUBLE_BLOCK_SIZE == 0 ? 0 : 1), prev_gradient_matrixes.depth);
	
	cuda_cross_correlation << <blocksPerGrid, threadsPerBlock >> > (prev_gradient_matrixes.data, prev_gradient_matrixes.cols_count, prev_gradient_matrixes.rows_count, prev_gradient_matrixes.pitch, gradients_device.cols_count, gradients_device.rows_count, filters_device.cols_count, gradients_device.depth, filters_device.cols_count - 1);
	cudacall(hipGetLastError());

	hipUnbindTexture(FiltersRef);
	hipUnbindTexture(MatrixesRef);
}

void ConvolutionalLayer::correct() {

	hipBindTexture2D(0, FiltersRef, gradients_device.data, FiltersRef.channelDesc, gradients_device.matrixes_size, gradients_device.depth, gradients_device.pitch);
	hipBindTexture2D(0, MatrixesRef, inputs_device.data, MatrixesRef.channelDesc, inputs_device.matrixes_size, inputs_device.depth, inputs_device.pitch);
	hipBindTexture2D(0, OutputsRef, outputs_devices.data, OutputsRef.channelDesc, outputs_devices.matrixes_size, outputs_devices.depth, outputs_devices.pitch);
	
	//It can be forced for larger images. This implementation appropriate only for MNIST.
	dim3 threadsPerBlock = dim3(DOUBLE_BLOCK_SIZE, DOUBLE_BLOCK_SIZE, 1);
	dim3 blocksPerGrid = dim3(filters_device.cols_count, filters_device.rows_count, filters_device.depth);

	cuda_correct_filters << <blocksPerGrid, threadsPerBlock, MAX_ELEMENTS_COUNT * sizeof(float) >> > (filters_device.data, filters_device.cols_count, filters_device.pitch, gradients_device.cols_count, gradients_device.rows_count, gradients_device.depth, inputs_device.cols_count, inputs_device.depth);
	cudacall(hipGetLastError())

	threadsPerBlock = BLOCK_SIZE;
	blocksPerGrid = dim3(gradients_device.depth / BLOCK_SIZE + (gradients_device.depth % BLOCK_SIZE == 0 ? 0 : 1));
	
	cuda_correct_biases << <blocksPerGrid, threadsPerBlock >> > (biases_device, gradients_device.depth, gradients_device.cols_count, gradients_device.rows_count);
	cudacall(hipGetLastError());

	hipUnbindTexture(FiltersRef);
	hipUnbindTexture(MatrixesRef);
	hipUnbindTexture(OutputsRef);
}

void ConvolutionalLayer::save_params(Hub& params_storage) {

	params_storage.set_params(filters_device);
	params_storage.set_params(biases_device, outputs_devices.depth);
}

void ConvolutionalLayer::freeMemory() {

	hipFree(filters_device.data);
	hipFree(gradients_device.data);
	hipFree(inputs_device.data);
	hipFree(biases_device);
}