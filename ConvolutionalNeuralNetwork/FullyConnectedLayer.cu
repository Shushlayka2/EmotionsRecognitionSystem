#include "hip/hip_runtime.h"
#include <malloc.h>
#include <iostream>

#include "Random.h"
#include "CustomException.h"
#include "FullyConnectedLayer.h"
#include "hip/hip_runtime.h"
#include ""

#define BLOCK_SIZE 256
#define DOUBLE_BLOCK_SIZE 32
#define LearningRate 0.001f

texture<float, 1, hipReadModeElementType> InputsRef;
texture<float, 1, hipReadModeElementType> GradientsRef;

__global__ void cuda_find_max(float* A, float* max, const int size)
{
	float max_val = _I32_MIN;
	for (int i = 0; i < size; i++)
	{
		max_val = __max(A[i], max_val);
	}
	max[0] = max_val;
}

__global__ void cuda_exp_vector_generate(float* A, float* B, float* max, const int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		B[idx] = __expf(A[idx] - max[0]);
	}
}

__global__ void cuda_softmax(float* A, float* max, const float log_val, const int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{

		A[idx] = __expf(A[idx] - max[0] - log_val);
	}
}

__global__ void cuda_set_gradients(float* gradients, float* outputs, const int num)
{
	int idx = threadIdx.x;
	gradients[idx] = outputs[idx];
	if (idx == num)
		gradients[idx] = outputs[idx] - 1;
}

__global__ void cuda_gr_to_der_mult(float* gradients, const int in_size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < in_size)
	{
		float inp_elem = tex1Dfetch(InputsRef, idx);
		gradients[idx] *= (inp_elem * (1 - inp_elem));
	}
}

__global__ void cuda_correct_weights(float* weights, const int inp_count, const int out_count)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idx < inp_count && idy < out_count)
	{
		weights[idy * inp_count + idx] -= LearningRate * tex1Dfetch(InputsRef, idx) * tex1Dfetch(GradientsRef, idy);
	}
}

__global__ void cuda_correct_biases(float* biases, const int out_count)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < out_count)
	{
		biases[idx] -= LearningRate * tex1Dfetch(GradientsRef, idx);
	}
}

FullyConnectedLayer::FullyConnectedLayer(int in_size, int out_size) {
	this->in_size = in_size;
	this->out_size = out_size;
	weights_device = set_normal_random(in_size * out_size, 1, weights_pitch);
	biases_device = set_repeatable_values(out_size, 0.01f);
	hipMalloc((void**)&gradients_device, out_size * sizeof(float));
	hipMalloc((void**)&outputs_device, out_size * sizeof(float));
	hipblasCreate(&handle);
}

float* FullyConnectedLayer::forward(float* prev_layer_data) {

	inputs_device = prev_layer_data;

	//test
	printf("Weights:\n");
	float* weights_host = new float[in_size * out_size];
	hipMemcpy(weights_host, weights_device, in_size * out_size * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < out_size; i++)
	{
		for (int j = 0; j < in_size; j++)
		{
			printf("%f ", weights_host[i * in_size + j]);
		}
		printf("\n");
	}
	printf("\n");
	free(weights_host);

	m_v_multiplication(weights_device, inputs_device, outputs_device, handle);

	//test
	printf("Outputs after mult:\n");
	float* outputs_host = new float[out_size];
	hipMemcpy(outputs_host, outputs_device, out_size * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < out_size; i++)
		printf("%f ", outputs_host[i]);
	printf("\n");

	add_biases(handle);

	//test
	printf("Outputs after biases adding:\n");
	outputs_host = new float[out_size];
	hipMemcpy(outputs_host, outputs_device, out_size * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < out_size; i++)
		printf("%f ", outputs_host[i]);
	printf("\n");

	activate_softmax(handle);

	//test
	printf("Outputs after activate:\n");
	outputs_host = new float[out_size];
	hipMemcpy(outputs_host, outputs_device, out_size * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < out_size; i++)
		printf("%f ", outputs_host[i]);
	printf("\n");
	free(outputs_host);

	return outputs_device;
}

void FullyConnectedLayer::backward(float* prev_layer_gradients) {
	
	hipBindTexture(0, InputsRef, inputs_device, in_size * sizeof(float));
	hipBindTexture(0, GradientsRef, gradients_device, out_size * sizeof(float));

	m_v_multiplication(weights_device, gradients_device, prev_layer_gradients, handle, HIPBLAS_OP_N);

	//test
	printf("Prev Gradients before der:\n");
	float* gradients_host = new float[in_size];
	hipMemcpy(gradients_host, prev_layer_gradients, in_size * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < in_size; i++)
		printf("%f ", gradients_host[i]);
	printf("\n");

	dim3 threadsPerBlock = BLOCK_SIZE;
	dim3 blocksPerGrid = in_size / BLOCK_SIZE + (in_size % BLOCK_SIZE == 0 ? 0 : 1);

	cuda_gr_to_der_mult << <blocksPerGrid, threadsPerBlock >> > (prev_layer_gradients, in_size);
	hipDeviceSynchronize();
	cudacall(hipGetLastError());

	printf("Prev Gradients after der:\n");
	hipMemcpy(gradients_host, prev_layer_gradients, in_size * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < in_size; i++)
		printf("%f ", gradients_host[i]);
	printf("\n");
	free(gradients_host);

	correct();

	hipUnbindTexture(InputsRef);
	hipUnbindTexture(GradientsRef);
}

void FullyConnectedLayer::correct() {
	
	dim3 threadsPerBlock = dim3(DOUBLE_BLOCK_SIZE, DOUBLE_BLOCK_SIZE);
	dim3 blocksPerGrid = dim3(in_size / DOUBLE_BLOCK_SIZE + (in_size % DOUBLE_BLOCK_SIZE == 0 ? 0 : 1),
		out_size / DOUBLE_BLOCK_SIZE + (out_size % DOUBLE_BLOCK_SIZE == 0 ? 0 : 1));
	cuda_correct_weights << <blocksPerGrid, threadsPerBlock >> > (weights_device, in_size, out_size);
	hipDeviceSynchronize();
	cudacall(hipGetLastError());

	//test
	printf("Corrected Weights:\n");
	float* weights_host = new float[in_size * out_size];
	hipMemcpy(weights_host, weights_device, in_size * out_size * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < out_size; i++)
	{
		for (int j = 0; j < in_size; j++)
		{
			printf("%f ", weights_host[i * in_size + j]);
		}
		printf("\n");
	}
	printf("\n");
	free(weights_host);

	threadsPerBlock = BLOCK_SIZE;
	blocksPerGrid = out_size / BLOCK_SIZE + (out_size % BLOCK_SIZE == 0 ? 0 : 1);
	cuda_correct_biases << <blocksPerGrid, threadsPerBlock >> > (biases_device, out_size);
	hipDeviceSynchronize();
	cudacall(hipGetLastError());

	//assert
	printf("Corrected Biases:\n");
	float* biases_host = new float[out_size];
	hipMemcpy(biases_host, biases_device, out_size * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < out_size; i++)
		printf("%f ", biases_host[i]);
	printf("\n");
	free(biases_host);
}

void FullyConnectedLayer::m_v_multiplication(float* matrix, float* vector, float* result_vector, hipblasHandle_t& handle, hipblasOperation_t trans) {
	float alpha = 1.0f, beta = 0.0f;
	cublascall(hipblasSgemv(handle, trans, in_size, out_size, &alpha, matrix, in_size, vector, 1, &beta, result_vector, 1));
}

void FullyConnectedLayer::add_biases(hipblasHandle_t& handle) {
	float alpha = 1.0f;
	cublascall(hipblasSaxpy(handle, out_size, &alpha, biases_device, 1, outputs_device, 1));
}

void FullyConnectedLayer::activate_softmax(hipblasHandle_t& handle) {
	float sum = 0.0f;
	float* max_device;
	float* helper_vector_device;
	dim3 threadsPerBlock = BLOCK_SIZE;
	dim3 blocksPerGrid = out_size / BLOCK_SIZE + (out_size % BLOCK_SIZE == 0 ? 0 : 1);
	
	hipMalloc((void**)&max_device, sizeof(float));
	hipMalloc((void**)&helper_vector_device, out_size * sizeof(float));
	cuda_find_max << <1, 1>> > (outputs_device, max_device, out_size);
	hipDeviceSynchronize();
	cudacall(hipGetLastError());

	cuda_exp_vector_generate << <blocksPerGrid, threadsPerBlock >> > (outputs_device, helper_vector_device, max_device, out_size);
	hipDeviceSynchronize();
	cudacall(hipGetLastError());

	cublascall(hipblasSasum(handle, out_size, helper_vector_device, 1, &sum));

	sum = log(sum);
	cuda_softmax << <blocksPerGrid, threadsPerBlock >> > (outputs_device, max_device, sum, out_size);
	hipDeviceSynchronize();
	cudacall(hipGetLastError());
	
	hipFree(helper_vector_device);
	hipFree(max_device);
}

void FullyConnectedLayer::set_gradients(int correct_result) {

	cuda_set_gradients << <1, 10 >> > (gradients_device, outputs_device, correct_result);
	hipDeviceSynchronize();
	cudacall(hipGetLastError());

	//test
	printf("Full Connected Backward:\n");
	printf("Gradients:\n");
	float* gradients_host = new float[out_size];
	hipMemcpy(gradients_host, gradients_device, out_size * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < out_size; i++)
		printf("%f ", gradients_host[i]);
	printf("\n");
	free(gradients_host);
}

float* FullyConnectedLayer::get_gradients() {
	return gradients_device;
}

int FullyConnectedLayer::get_result() {
	
	int max_index;
	cublascall(hipblasIsamax(handle, out_size, outputs_device, 1, &max_index));
	return max_index - 1;
}

void FullyConnectedLayer::freeMemory() {
	hipFree(inputs_device);
	hipFree(outputs_device);
	hipFree(gradients_device);
	hipFree(weights_device);
	hipFree(biases_device);
	hipblasDestroy(handle);
}