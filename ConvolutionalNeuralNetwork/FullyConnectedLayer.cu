#include "hip/hip_runtime.h"
#include <malloc.h>
#include <iostream>

#include "Random.h"
#include "CustomException.h"
#include "FullyConnectedLayer.h"
#include "hip/hip_runtime.h"
#include ""

#define BLOCK_SIZE 256

__global__ void cuda_find_max(float* A, float* max, const int size)
{
	float max_val = _I32_MIN;
	for (int i = 0; i < size; i++)
	{
		max_val = __max(A[i], max_val);
	}
	max[0] = max_val;
}

__global__ void cuda_exp_vector_generate(float* A, float* B, float* max, const int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		B[idx] = __expf(A[idx] - max[0]);
	}
}

__global__ void cuda_softmax(float* A, float* max, const float log_val, const int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{

		A[idx] = __expf(A[idx] - max[0] - log_val);
	}
}

__global__ void cuda_set_gradients(float* gradients, float* outputs, const int num)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	gradients[idx] = outputs[idx];
	if (idx == num)
		gradients[idx] = 1 - outputs[idx];
}

float find_max(float* arr, int size);

FullyConnectedLayer::FullyConnectedLayer(int in_size, int out_size) {
	this->in_size = in_size;
	this->out_size = out_size;
	weights_device = set_normal_random(in_size * out_size, 1, weights_pitch);
	biases_device = set_normal_random(out_size, 1, weights_pitch);
	hipMalloc((void**)&gradients_device, out_size * sizeof(float));
	hipMalloc((void**)&outputs_device, out_size * sizeof(float));
	hipblasCreate(&handle);
}

float* FullyConnectedLayer::forward(float* prev_layer_data) {

	inputs_device = prev_layer_data;

	m_v_multiplication(weights_device, inputs_device, outputs_device, handle);
	add_biases(handle);
	activate_softmax(handle);

	return outputs_device;
}

void FullyConnectedLayer::backward(float* prev_layer_gradients) {

	m_v_multiplication(weights_device, gradients_device, prev_layer_gradients, handle, HIPBLAS_OP_N);
}

void FullyConnectedLayer::m_v_multiplication(float* matrix, float* vector, float* result_vector, hipblasHandle_t& handle, hipblasOperation_t trans) {
	float alpha = 1.0f, beta = 0.0f;
	cublascall(hipblasSgemv(handle, trans, in_size, out_size, &alpha, matrix, in_size, vector, 1, &beta, result_vector, 1));
}

void FullyConnectedLayer::add_biases(hipblasHandle_t& handle) {
	float alpha = 1.0f;
	cublascall(hipblasSaxpy(handle, out_size, &alpha, biases_device, 1, outputs_device, 1));
}

void FullyConnectedLayer::activate_softmax(hipblasHandle_t& handle) {
	float sum = 0.0f;
	float* max_device;
	float* helper_vector_device;
	dim3 threadsPerBlock = BLOCK_SIZE;
	dim3 blocksPerGrid = out_size / BLOCK_SIZE + (out_size % BLOCK_SIZE == 0 ? 0 : 1);
	
	hipMalloc((void**)&max_device, sizeof(float));
	hipMalloc((void**)&helper_vector_device, out_size * sizeof(float));
	cuda_find_max << <1, 1>> > (outputs_device, max_device, out_size);
	hipDeviceSynchronize();
	cudacall(hipGetLastError());

	cuda_exp_vector_generate << <blocksPerGrid, threadsPerBlock >> > (outputs_device, helper_vector_device, max_device, out_size);
	cudacall(hipGetLastError());

	cublascall(hipblasSasum(handle, out_size, helper_vector_device, 1, &sum));

	sum = log(sum);
	cuda_softmax << <blocksPerGrid, threadsPerBlock >> > (outputs_device, max_device, sum, out_size);
	cudacall(hipGetLastError());
	
	hipFree(helper_vector_device);
	hipFree(max_device);
}

void FullyConnectedLayer::set_gradients(int correct_result) {

	cuda_set_gradients << <1, 10 >> > (gradients_device, outputs_device, correct_result);
}

float* FullyConnectedLayer::get_gradients() {
	return gradients_device;
}

void FullyConnectedLayer::freeMemory() {
	hipFree(inputs_device);
	hipFree(outputs_device);
	hipFree(gradients_device);
	hipFree(weights_device);
	hipFree(biases_device);
	hipblasDestroy(handle);
}