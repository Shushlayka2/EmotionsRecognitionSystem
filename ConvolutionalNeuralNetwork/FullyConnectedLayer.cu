#include "hip/hip_runtime.h"
#include <iostream>

#include "Hub.h"
#include "Random.h"
#include "CustomException.h"
#include "FullyConnectedLayer.h"
#include "hip/hip_runtime.h"
#include ""

#define Y 0.0001f
#define BLOCK_SIZE 512
#define DOUBLE_BLOCK_SIZE 32
#define LearningRate 0.005f

texture<float, 1, hipReadModeElementType> InputsRef;
texture<float, 1, hipReadModeElementType> GradientsRef;

__global__ void cuda_find_max(float* A, float* max, const int size)
{
	float max_val = _I32_MIN;
	for (int i = 0; i < size; i++)
	{
		max_val = __max(A[i], max_val);
	}
	max[0] = max_val;
}

__global__ void cuda_exp_vector_generate(float* A, float* sum, float* max, const int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		extern __shared__ float s_parts[];
		s_parts[idx] = __expf(A[idx] - max[0]);
		__syncthreads();
		if (idx == 0) {
			float sum_local = 0.0f;
			for (int i = 0; i < size; i++)
				sum_local += s_parts[i];
			sum[0] = log(sum_local);
		}

	}
}

__global__ void cuda_softmax(float* A, float* max, float* log_val, const int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		A[idx] = __expf(A[idx] - max[0] - log_val[0]);
	}
}

__global__ void cuda_sigmoid(float* outputs, const int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		outputs[idx] = 1 / (1 + __expf(-1 * outputs[idx]));
	}
}

__global__ void cuda_set_gradients(float* gradients, float* outputs, const int num)
{
	int idx = threadIdx.x;
	gradients[idx] = outputs[idx];
	if (idx == num)
		gradients[idx] = outputs[idx] - 1;
}

__global__ void cuda_gr_to_der_mult(float* gradients, const int in_size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < in_size)
	{
		float inp_elem = tex1Dfetch(InputsRef, idx);
		gradients[idx] *= (inp_elem * (1 - inp_elem));
	}
}

__global__ void cuda_correct_weights(float* weights, const int inp_count, const int out_count)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	if (idx < inp_count && idy < out_count)
	{
		weights[idy * inp_count + idx] = (1 - Y) * weights[idy * inp_count + idx] - LearningRate * tex1Dfetch(InputsRef, idx) * tex1Dfetch(GradientsRef, idy);
	}
}

__global__ void cuda_correct_biases(float* biases, const int out_count)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < out_count)
	{
		biases[idx] = (1 - Y) * biases[idx] - LearningRate * tex1Dfetch(GradientsRef, idx);
	}
}

FullyConnectedLayer::FullyConnectedLayer(int in_size, int out_size, Hub& params_storage, ActivationType type) : inputs_device(nullptr) {
	
	network_error = 0.0f;
	this->in_size = in_size;
	this->out_size = out_size;
	this->type = type;
	hipMalloc((void**)&gradients_device, out_size * sizeof(float));
	hipMalloc((void**)&outputs_device, out_size * sizeof(float));
	hipMalloc((void**)&sum, sizeof(float));
	hipMalloc((void**)&max_device, sizeof(float));
	hipblasCreate(&handle);

	if (params_storage.get_status() == Status::Training)
	{
		weights_device = set_normal_random(in_size * out_size, 1, weights_pitch, 2 / ((float)(in_size + out_size)), false);
		biases_device = set_repeatable_values(out_size, 0.01f);
	}
	else
	{
		weights_device = params_storage.get_params(in_size * out_size);
		biases_device = params_storage.get_params(out_size);
	}
}

float* FullyConnectedLayer::forward(float* prev_layer_data) {

	inputs_device = prev_layer_data;
	m_v_multiplication(weights_device, inputs_device, outputs_device, handle);
	add_biases(handle);
	activate(handle);
	return outputs_device;
}

void FullyConnectedLayer::backward(float* prev_layer_gradients) {
	
	hipBindTexture(0, InputsRef, inputs_device, in_size * sizeof(float));
	hipBindTexture(0, GradientsRef, gradients_device, out_size * sizeof(float));

	m_v_multiplication(weights_device, gradients_device, prev_layer_gradients, handle, HIPBLAS_OP_N);

	dim3 threadsPerBlock = BLOCK_SIZE;
	dim3 blocksPerGrid = in_size / BLOCK_SIZE + (in_size % BLOCK_SIZE == 0 ? 0 : 1);

	cuda_gr_to_der_mult << <blocksPerGrid, threadsPerBlock >> > (prev_layer_gradients, in_size);
	cudacall(hipGetLastError());

	correct();

	hipUnbindTexture(InputsRef);
	hipUnbindTexture(GradientsRef);
}

void FullyConnectedLayer::correct() {
	
	dim3 threadsPerBlock = dim3(DOUBLE_BLOCK_SIZE, DOUBLE_BLOCK_SIZE);
	dim3 blocksPerGrid = dim3(in_size / DOUBLE_BLOCK_SIZE + (in_size % DOUBLE_BLOCK_SIZE == 0 ? 0 : 1),
		out_size / DOUBLE_BLOCK_SIZE + (out_size % DOUBLE_BLOCK_SIZE == 0 ? 0 : 1));
	cuda_correct_weights << <blocksPerGrid, threadsPerBlock >> > (weights_device, in_size, out_size);
	cudacall(hipGetLastError());

	threadsPerBlock = BLOCK_SIZE;
	blocksPerGrid = out_size / BLOCK_SIZE + (out_size % BLOCK_SIZE == 0 ? 0 : 1);
	cuda_correct_biases << <blocksPerGrid, threadsPerBlock >> > (biases_device, out_size);
	cudacall(hipGetLastError());
}

void FullyConnectedLayer::m_v_multiplication(float* matrix, float* vector, float* result_vector, hipblasHandle_t& handle, hipblasOperation_t trans) {
	
	float alpha = 1.0f, beta = 0.0f;
	cublascall(hipblasSgemv(handle, trans, in_size, out_size, &alpha, matrix, in_size, vector, 1, &beta, result_vector, 1));
}

void FullyConnectedLayer::add_biases(hipblasHandle_t& handle) {
	
	float alpha = 1.0f;
	cublascall(hipblasSaxpy(handle, out_size, &alpha, biases_device, 1, outputs_device, 1));
}

void FullyConnectedLayer::activate(hipblasHandle_t& handle) {
	
	if (type == ActivationType::Softmax)
		activate_softmax(handle);
	else
		activate_sigmoid();
}

void FullyConnectedLayer::activate_sigmoid() {
	
	dim3 threadsPerBlock = BLOCK_SIZE;
	dim3 blocksPerGrid = out_size / BLOCK_SIZE + (out_size % BLOCK_SIZE == 0 ? 0 : 1);
	cuda_sigmoid << <blocksPerGrid, threadsPerBlock >> > (outputs_device, out_size);
	cudacall(hipGetLastError());
}

void FullyConnectedLayer::activate_softmax(hipblasHandle_t& handle) {
	
	dim3 threadsPerBlock = BLOCK_SIZE;
	dim3 blocksPerGrid = out_size / BLOCK_SIZE + (out_size % BLOCK_SIZE == 0 ? 0 : 1);

	cuda_find_max << <1, 1>> > (outputs_device, max_device, out_size);
	cudacall(hipGetLastError());

	cuda_exp_vector_generate << <blocksPerGrid, threadsPerBlock, out_size * sizeof(float) >> > (outputs_device, sum, max_device, out_size);
	cudacall(hipGetLastError());

	cuda_softmax << <blocksPerGrid, threadsPerBlock >> > (outputs_device, max_device, sum, out_size);
	cudacall(hipGetLastError());
}

void FullyConnectedLayer::set_gradients(int correct_result) {

	cuda_set_gradients << <1, out_size >> > (gradients_device, outputs_device, correct_result);
	cudacall(hipGetLastError());
}

int FullyConnectedLayer::get_result() {
	
	int max_index;
	cublascall(hipblasIsamax(handle, out_size, outputs_device, 1, &max_index));
	return max_index - 1;
}

void FullyConnectedLayer::calc_error(int correct_result) {

	float local_error;
	hipMemcpy(&local_error, outputs_device + correct_result, sizeof(float), hipMemcpyDeviceToHost);
	network_error -= log(local_error);
}

float FullyConnectedLayer::get_common_error(const int set_size) {

	float test = network_error / set_size;
	network_error = 0.0f;
	return test;
}

void FullyConnectedLayer::save_params(Hub& params_storage) {

	params_storage.set_params(weights_device, in_size * out_size);
	params_storage.set_params(biases_device, out_size);
}

void FullyConnectedLayer::freeMemory() {
	
	hipFree(sum);
	hipFree(max_device);
	hipFree(gradients_device);
	hipFree(weights_device);
	hipFree(biases_device);
	hipFree(outputs_device);
	hipblasDestroy(handle);
}