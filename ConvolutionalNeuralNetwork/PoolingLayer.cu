#include "hip/hip_runtime.h"
#include <iostream>

#include "PoolingLayer.h"
#include "CustomException.h"
#include "hip/hip_runtime.h"
#include ""

#define BLOCK_SIZE 32

texture<float, 2> InputMatrixesRef;
texture<float, 2> GradientMatrixesRef;

__global__ void cuda_pooling(float* result, float* prev_gradients, size_t gr_pitch, const int cols, const int rows, size_t fm_pitch, const int feature_map_cols, const int filter_size)
{
	int block_x = blockDim.x * blockIdx.x + threadIdx.x;
	int block_y = blockDim.y * blockIdx.y + threadIdx.y;
	int block_z = blockDim.z * blockIdx.z + threadIdx.z;

	int x = block_x * filter_size; 
	int y = block_y * filter_size;

	if (x < cols && y < rows)
	{
		y *= cols;
		int filter_upper_position = y + x;
		int filter_right_border = __min(x + filter_size - 1, cols - 1) - x;
		int filter_bottom_position = __min(filter_upper_position + cols * (filter_size - 1), x + cols * (rows - 1));

		float* prev_gradients_start = (float*)((char*)prev_gradients + block_z * gr_pitch);

		float max_val = 0.0f;
		for (int i = filter_upper_position; i <= filter_bottom_position; i += cols)
		{
			for (int j = 0; j <= filter_right_border; j++)
			{
				float element = tex2D(InputMatrixesRef, i + j, block_z);
				max_val = __max(element, max_val);
				prev_gradients[i + j] = max_val;
			}
		}

		for (int i = filter_upper_position; i <= filter_bottom_position; i += cols)
		{
			for (int j = 0; j <= filter_right_border; j++)
			{
				prev_gradients[i + j] = prev_gradients[i + j] == max_val;
			}
		}

		float* feature_map_matrix_start = (float*)((char*)result + block_z * fm_pitch);
		int feature_map_position = block_y * feature_map_cols + block_x;

		feature_map_matrix_start[feature_map_position] = max_val;
	}
}

__global__ void cuda_generate_gradients(float* prev_gradients, size_t prev_gr_pitch, const int cols, const int rows, const int cur_gr_cols, const int filter_size)
{
	int block_x = blockDim.x * blockIdx.x + threadIdx.x;
	int block_y = blockDim.y * blockIdx.y + threadIdx.y;
	int block_z = blockDim.z * blockIdx.z + threadIdx.z;

	int x = block_x * filter_size;
	int y = block_y * filter_size;

	if (x < cols && y < rows)
	{
		y *= cols;
		int filter_upper_position = y + x;
		int filter_right_border = __min(x + filter_size - 1, cols - 1) - x;
		int filter_bottom_position = __min(filter_upper_position + cols * (filter_size - 1), x + cols * (rows - 1));
		float* prev_gradients_start = (float*)((char*)prev_gradients + block_z * prev_gr_pitch);

		for (int i = filter_upper_position; i <= filter_bottom_position; i += cols)
		{
			for (int j = 0; j <= filter_right_border; j++)
			{
				float element = tex2D(GradientMatrixesRef, block_y * cur_gr_cols + block_x, block_z);
				prev_gradients_start[i + j] *= element;
			}
		}
	}
}

PoolingLayer::PoolingLayer(const int filter_size, const int gradients_size, const int gradients_depth) {
	this->filter_size = filter_size;
	gradients_device = MatrixBlock(gradients_size, gradients_size, gradients_depth);
	hipMallocPitch((void**)&gradients_device.data, &gradients_device.pitch, gradients_device.matrixes_size, gradients_device.depth);
}

MatrixBlock& PoolingLayer::forward(MatrixBlock& input_matrixes, MatrixBlock& prev_gradient_matrixes) {
	
	inputs_device = input_matrixes;
	unsigned int pooled_feature_map_cols = inputs_device.cols_count / filter_size + (inputs_device.cols_count % filter_size == 0 ? 0 : 1);
	unsigned int pooled_feature_map_rows = inputs_device.rows_count / filter_size + (inputs_device.rows_count % filter_size == 0 ? 0 : 1);

	outputs_devices = MatrixBlock(pooled_feature_map_rows, pooled_feature_map_cols, inputs_device.depth);
	hipMallocPitch((void**)&outputs_devices.data, &outputs_devices.pitch, outputs_devices.matrixes_size * sizeof(float), outputs_devices.depth);

	hipBindTexture2D(0, InputMatrixesRef, inputs_device.data, InputMatrixesRef.channelDesc, inputs_device.matrixes_size, inputs_device.depth, inputs_device.pitch);
	
	dim3 threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 blocksPerGrid = dim3(pooled_feature_map_cols / BLOCK_SIZE + (pooled_feature_map_cols % BLOCK_SIZE == 0 ? 0 : 1), pooled_feature_map_rows / BLOCK_SIZE + (pooled_feature_map_rows % BLOCK_SIZE == 0 ? 0 : 1), inputs_device.depth);
	cuda_pooling << <blocksPerGrid, threadsPerBlock >> > (outputs_devices.data, prev_gradient_matrixes.data, prev_gradient_matrixes.pitch, input_matrixes.cols_count, input_matrixes.rows_count, outputs_devices.pitch, outputs_devices.cols_count, filter_size);
	hipDeviceSynchronize();
	cudacall(hipGetLastError());

	hipUnbindTexture(InputMatrixesRef);

	return outputs_devices;
}

void PoolingLayer::backward(MatrixBlock& prev_gradient_matrixes) {
	
	dim3 threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 blocksPerGrid = dim3(gradients_device.cols_count / BLOCK_SIZE + (gradients_device.cols_count % BLOCK_SIZE == 0 ? 0 : 1), gradients_device.rows_count / BLOCK_SIZE + (gradients_device.rows_count % BLOCK_SIZE == 0 ? 0 : 1), gradients_device.depth);
	hipBindTexture2D(0, GradientMatrixesRef, gradients_device.data, GradientMatrixesRef.channelDesc, gradients_device.matrixes_size, gradients_device.depth, gradients_device.pitch);
	
	cuda_generate_gradients << <blocksPerGrid, threadsPerBlock >> > (prev_gradient_matrixes.data, prev_gradient_matrixes.pitch, prev_gradient_matrixes.cols_count, prev_gradient_matrixes.rows_count, gradients_device.cols_count, filter_size);
	hipDeviceSynchronize();
	cudacall(hipGetLastError());

	hipUnbindTexture(InputMatrixesRef);
}

void PoolingLayer::freeMemory() {
	hipFree(inputs_device.data);
	hipFree(gradients_device.data);
	hipFree(outputs_devices.data);
}
