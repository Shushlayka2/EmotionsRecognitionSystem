#include "hip/hip_runtime.h"
#include <iostream>

#include "PoolingLayer.h"
#include "CustomException.h"
#include "hip/hip_runtime.h"
#include ""

#define BLOCK_SIZE 32

texture<float, 2> InputMatrixesRef;
texture<float, 2> GradientMatrixesRef;

__global__ void cuda_pooling(float* result, float* prev_gradients, size_t gr_pitch, const int cols, const int rows, size_t fm_pitch, const int feature_map_cols, const int filter_size)
{
	int block_x = blockDim.x * blockIdx.x + threadIdx.x;
	int block_y = blockDim.y * blockIdx.y + threadIdx.y;
	int block_z = blockDim.z * blockIdx.z + threadIdx.z;

	int x = block_x * filter_size; 
	int y = block_y * filter_size;

	if (x < cols && y < rows)
	{
		y *= cols;
		int filter_upper_position = y + x;
		int filter_right_border = __min(x + filter_size - 1, cols - 1) - x;
		int filter_bottom_position = __min(filter_upper_position + cols * (filter_size - 1), x + cols * (rows - 1));

		float* prev_gradients_start = (float*)((char*)prev_gradients + block_z * gr_pitch);

		float max_val = _I32_MIN;
		int max_i, max_j;
		for (int i = filter_upper_position; i <= filter_bottom_position; i += cols)
		{
			for (int j = 0; j <= filter_right_border; j++)
			{
				float element = tex2D(InputMatrixesRef, i + j, block_z);
				(element > max_val) ? (max_val = element, max_i = i, max_j = j) : (max_val);
				prev_gradients_start[i + j] = 0;
			}
		}

		prev_gradients_start[max_i + max_j] = 1.0f;
		float* feature_map_matrix_start = (float*)((char*)result + block_z * fm_pitch);
		int feature_map_position = block_y * feature_map_cols + block_x;

		feature_map_matrix_start[feature_map_position] = max_val;
	}
}

__global__ void cuda_generate_gradients(float* prev_gradients, size_t prev_gr_pitch, const int cols, const int rows, const int cur_gr_cols, const int filter_size)
{
	int block_x = blockDim.x * blockIdx.x + threadIdx.x;
	int block_y = blockDim.y * blockIdx.y + threadIdx.y;
	int block_z = blockDim.z * blockIdx.z + threadIdx.z;

	int x = block_x * filter_size;
	int y = block_y * filter_size;

	if (x < cols && y < rows)
	{
		y *= cols;
		int filter_upper_position = y + x;
		int filter_right_border = __min(x + filter_size - 1, cols - 1) - x;
		int filter_bottom_position = __min(filter_upper_position + cols * (filter_size - 1), x + cols * (rows - 1));
		float* prev_gradients_start = (float*)((char*)prev_gradients + block_z * prev_gr_pitch);

		float element = tex2D(GradientMatrixesRef, block_y * cur_gr_cols + block_x, block_z);
		for (int i = filter_upper_position; i <= filter_bottom_position; i += cols)
		{
			for (int j = 0; j <= filter_right_border; j++)
			{	
				prev_gradients_start[i + j] *= element;
			}
		}
	}
}

PoolingLayer::PoolingLayer(const int filter_size, const int outputs_size, const int outputs_depth) {
	
	this->filter_size = filter_size;
	gradients_device = Tensor(outputs_size, outputs_size, outputs_depth);
	hipMallocPitch((void**)&gradients_device.data, &gradients_device.pitch, gradients_device.matrixes_size * sizeof(float), gradients_device.depth);

	outputs_devices = Tensor(outputs_size, outputs_size, outputs_depth);
	hipMallocPitch((void**)&outputs_devices.data, &outputs_devices.pitch, outputs_devices.matrixes_size * sizeof(float), outputs_devices.depth);
}

Tensor& PoolingLayer::forward(Tensor& input_matrixes, Tensor& prev_gradient_matrixes) {
	
	inputs_device = input_matrixes;

	hipBindTexture2D(0, InputMatrixesRef, inputs_device.data, InputMatrixesRef.channelDesc, inputs_device.matrixes_size, inputs_device.depth, inputs_device.pitch);
	
	dim3 threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 blocksPerGrid = dim3(outputs_devices.cols_count / BLOCK_SIZE + (outputs_devices.cols_count % BLOCK_SIZE == 0 ? 0 : 1), outputs_devices.rows_count / BLOCK_SIZE + (outputs_devices.rows_count % BLOCK_SIZE == 0 ? 0 : 1), outputs_devices.depth);
	cuda_pooling << <blocksPerGrid, threadsPerBlock >> > (outputs_devices.data, prev_gradient_matrixes.data, prev_gradient_matrixes.pitch, input_matrixes.cols_count, input_matrixes.rows_count, outputs_devices.pitch, outputs_devices.cols_count, filter_size);
	cudacall(hipGetLastError());

	hipUnbindTexture(InputMatrixesRef);

	return outputs_devices;
}

void PoolingLayer::backward(Tensor& prev_gradient_matrixes) {

	dim3 threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
	dim3 blocksPerGrid = dim3(gradients_device.cols_count / BLOCK_SIZE + (gradients_device.cols_count % BLOCK_SIZE == 0 ? 0 : 1), gradients_device.rows_count / BLOCK_SIZE + (gradients_device.rows_count % BLOCK_SIZE == 0 ? 0 : 1), gradients_device.depth);
	hipBindTexture2D(0, GradientMatrixesRef, gradients_device.data, GradientMatrixesRef.channelDesc, gradients_device.matrixes_size, gradients_device.depth, gradients_device.pitch);

	cuda_generate_gradients << <blocksPerGrid, threadsPerBlock >> > (prev_gradient_matrixes.data, prev_gradient_matrixes.pitch, prev_gradient_matrixes.cols_count, prev_gradient_matrixes.rows_count, gradients_device.cols_count, filter_size);
	cudacall(hipGetLastError());

	hipUnbindTexture(InputMatrixesRef);
}

void PoolingLayer::freeMemory() {
	
	hipFree(gradients_device.data);
	hipFree(outputs_devices.data);
}
