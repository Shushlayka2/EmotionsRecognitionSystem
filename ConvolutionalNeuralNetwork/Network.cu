#include "hip/hip_runtime.h"
#include "Network.h"
#include "CustomException.h"
#include "PoolingLayer.h"
#include "FullyConnectedLayer.h"
#include "hip/hip_runtime.h"
#include ""

#define BLOCK_SIZE 256

__global__ void cuda_normalize(float* inputs, const int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		inputs[idx] /= 255;
	}
}

Network::Network(ConfigHandler& configurationHandler) {
	this->configurationHandler = configurationHandler;
	this->image_size = configurationHandler.Value("image_size");
	this->filter_size = configurationHandler.Value("filter_size");
	this->amount_of_filters = configurationHandler.Value("amount_of_filters");
	this->pooling_filters_size = configurationHandler.Value("pooling_filters_size");
	this->convolutional_layers_count = configurationHandler.Value("convolution_layers_count");
	this->fully_connected_layers_count = configurationHandler.Value("fully_connected_layers_count");
	this->fully_connected_layers_neurons_count = configurationHandler.VectorValue("fully_connected_layers_neurons_count");
	init_layers();
}

void Network::run() {
	
	Tensor& current_matrix_block = inputs_device;

	for (int i = 0; i < convolutional_layers_count; i++)
	{
		current_matrix_block = convolutionalLayers[i].forward(current_matrix_block);
		current_matrix_block = poolingLayers[i].forward(current_matrix_block, convolutionalLayers[i].gradients_device);
	}

	float* current_input_vector;
	hipMalloc((void**)&current_input_vector, current_matrix_block.matrixes_size * current_matrix_block.depth * sizeof(float));
	hipMemcpy2D(current_input_vector, current_matrix_block.matrixes_size * sizeof(float), current_matrix_block.data, current_matrix_block.pitch,
		current_matrix_block.matrixes_size * sizeof(float), current_matrix_block.depth, hipMemcpyDeviceToDevice);

	//test
	/*printf("Fully Connected Forward:\n");
	printf("Inputs:\n");
	float* inputs_host = new float[current_matrix_block.matrixes_size * current_matrix_block.depth];
	hipMemcpy(inputs_host, current_input_vector, current_matrix_block.matrixes_size * current_matrix_block.depth * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < current_matrix_block.matrixes_size * current_matrix_block.depth; i++)
		printf("%f ", inputs_host[i]);
	printf("\n");*/

	for (int i = 0; i < fully_connected_layers_count; i++)
	{
		current_input_vector = fullyConnectedLayers[i].forward(current_input_vector);
	}
}

void Network::correct(int correct_result) {

	fullyConnectedLayers[fully_connected_layers_count - 1].set_gradients(correct_result);
	for (int i = fully_connected_layers_count - 1; i > 0; i--)
	{
		fullyConnectedLayers[i].backward(fullyConnectedLayers[i - 1].get_gradients());
	}

	Tensor cur_gradients_mb = poolingLayers[convolutional_layers_count - 1].gradients_device;
	float* first_pl_gr_vector_device;
	hipMalloc((void**)&first_pl_gr_vector_device, cur_gradients_mb.matrixes_size * cur_gradients_mb.depth * sizeof(float));
	fullyConnectedLayers[0].backward(first_pl_gr_vector_device);

	hipMemcpy2D(cur_gradients_mb.data, cur_gradients_mb.pitch, first_pl_gr_vector_device, cur_gradients_mb.matrixes_size * sizeof(float),
		cur_gradients_mb.matrixes_size * sizeof(float), cur_gradients_mb.depth, hipMemcpyDeviceToDevice);

	for (int i = convolutional_layers_count - 1; i > 0; i--)
	{
		poolingLayers[i].backward(convolutionalLayers[i].gradients_device);
		convolutionalLayers[i].correct();
		convolutionalLayers[i].backward(poolingLayers[i - 1].gradients_device);
	}
	poolingLayers[0].backward(convolutionalLayers[0].gradients_device);
	convolutionalLayers[0].correct();
}

void Network::init_layers() {

	int depth = 1;
	int prev_layer_neurons_count = image_size;
	for (int i = 0; i < convolutional_layers_count; i++)
	{
		depth *= amount_of_filters;

		prev_layer_neurons_count = prev_layer_neurons_count - filter_size + 1;
		ConvolutionalLayer conv_layer = ConvolutionalLayer(filter_size, amount_of_filters, prev_layer_neurons_count, depth);

		prev_layer_neurons_count = prev_layer_neurons_count / 2 + (prev_layer_neurons_count % 2 == 0 ? 0 : 1);
		PoolingLayer pooling_layer = PoolingLayer(pooling_filters_size, prev_layer_neurons_count, depth);
		
		convolutionalLayers.push_back(conv_layer);
		poolingLayers.push_back(pooling_layer);
	}

	prev_layer_neurons_count *= prev_layer_neurons_count * depth;

	for (int i = 0; i < fully_connected_layers_count - 1; i++)
	{
		int next_layer_neurons_count = fully_connected_layers_neurons_count[i];
		FullyConnectedLayer fullyconnected_layer = FullyConnectedLayer(prev_layer_neurons_count, next_layer_neurons_count);
		fullyConnectedLayers.push_back(fullyconnected_layer);
		prev_layer_neurons_count = next_layer_neurons_count;
	}
	int next_layer_neurons_count = fully_connected_layers_neurons_count[fully_connected_layers_count - 1];
	FullyConnectedLayer fullyconnected_layer = FullyConnectedLayer(prev_layer_neurons_count, next_layer_neurons_count, ActivationType::Softmax);
	fullyConnectedLayers.push_back(fullyconnected_layer);
}

void Network::set_inputs(Tensor& image_matrix_block) {

	inputs_device = image_matrix_block;
	float* data_host = image_matrix_block.data;
	hipMallocPitch((void**)&inputs_device.data, &inputs_device.pitch, inputs_device.matrixes_size * sizeof(float), inputs_device.depth);
	hipMemcpy2D(inputs_device.data, inputs_device.pitch, data_host, inputs_device.matrixes_size * sizeof(float), inputs_device.matrixes_size * sizeof(float), inputs_device.depth, hipMemcpyHostToDevice);
	dim3 threadsPerBlock = BLOCK_SIZE;
	dim3 blocksPerGrid = inputs_device.matrixes_size / BLOCK_SIZE + (inputs_device.matrixes_size % BLOCK_SIZE == 0 ? 0 : 1);
	cuda_normalize << <blocksPerGrid, threadsPerBlock >> > (inputs_device.data, inputs_device.matrixes_size);
}

int Network::get_result() {
	return fullyConnectedLayers[fully_connected_layers_count - 1].get_result();
}

void Network::free_inputs() {

	hipFree(inputs_device.data);
}